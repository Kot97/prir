#include <iostream>
#include <builtin_types.h>
#include "kernels/disarium_number.cuh"

const unsigned int NUMBERS_COUNT = 12;

unsigned long *allocateArrayOnGPU(unsigned long elementsCount, size_t elementSize);

void synchronizeKernel();

void transferDataFromGPU(unsigned long *generatedNumbersGPU, unsigned long *generatedNumbersCPU);

void printResult(const unsigned long *generatedNumbersCPU);

int main(int argc, char **argv) {
    unsigned long *generatedNumbersGPU = allocateArrayOnGPU(NUMBERS_COUNT, sizeof(unsigned long));
    unsigned long *generatedNumbersCPU = new unsigned long[NUMBERS_COUNT];

    //todo update thread/blocks calculator
    unsigned long threadNum = 1;
    unsigned long blocksNum = 1;
    std::cout << "block num: " << blocksNum << std::endl << "threads count: " << threadNum << std::endl;
    generateDisariumNumbers <<< blocksNum, threadNum >>>(generatedNumbersGPU, NUMBERS_COUNT);

    synchronizeKernel();
    transferDataFromGPU(generatedNumbersGPU, generatedNumbersCPU);
    printResult(generatedNumbersCPU);

    hipFree(generatedNumbersGPU);
    delete[] generatedNumbersCPU;

    return 0;
}

void printResult(const unsigned long *generatedNumbersCPU) {
    for (unsigned int i = 0; i < NUMBERS_COUNT; i++)
        std::cout << generatedNumbersCPU[i] << std::endl;
}

void transferDataFromGPU(unsigned long *generatedNumbersGPU, unsigned long *generatedNumbersCPU) {
    hipError_t errorCode = hipMemcpy(generatedNumbersCPU, generatedNumbersGPU, sizeof(unsigned long) * NUMBERS_COUNT,
                                       hipMemcpyDeviceToHost);
    if (errorCode != hipSuccess) {
        std::cout << "error during transfer data from gpu " << hipGetErrorName(errorCode)
                  << std::endl;
        exit(EXIT_FAILURE);
    }
}

void synchronizeKernel() {
    hipError_t errorCode = hipDeviceSynchronize();
    if (errorCode != hipSuccess) {
        std::cout << "error during Device Synchronize: " << hipGetErrorName(errorCode)
                  << std::endl;
        exit(EXIT_FAILURE);
    }
}

unsigned long *allocateArrayOnGPU(const unsigned long elementsCount, const size_t elementSize) {
    unsigned long *table_addr;
    hipError_t errorCode = hipMalloc((void **) &table_addr, elementsCount * elementSize);
    if (errorCode != hipSuccess) {
        std::cout << "error during alloc memory for digest on GPU error code: " << hipGetErrorName(errorCode)
                  << std::endl;
        exit(EXIT_FAILURE);
    }
    return table_addr;
}