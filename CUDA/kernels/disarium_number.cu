
#include <hip/hip_runtime.h>
__device__ unsigned long countDigits(unsigned long number);

__device__ bool isNumberDisarium(unsigned long number);

__device__ unsigned long pow(unsigned long x,unsigned long n);

__device__ void addResult(unsigned long *generatedNumbersGPU, unsigned long result);

__global__ void generateDisariumNumbers(unsigned long *generatedNumbersGPU, const unsigned long NUMBERS_COUNT) {
    unsigned long i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < NUMBERS_COUNT) {
        if (isNumberDisarium(i))
            addResult(generatedNumbersGPU, i);
        i += blockDim.x*gridDim.x;
    }
}

__device__ void addResult(unsigned long *generatedNumbersGPU, unsigned long result) {

}

__device__ bool isNumberDisarium(unsigned long number) {
    unsigned long sum = 0, temp = number;
    unsigned long digitsCount = countDigits(number);
    while (temp) {
        sum += pow(temp % 10, digitsCount--);
        temp /= 10;
    }
    return sum == number;
}

__device__ unsigned long countDigits(unsigned long number) {
    unsigned int digits_count = 0;
    while (number) {
        number /= 10;
        digits_count++;
    }
    return digits_count;
}

__device__ unsigned long pow(unsigned long x, unsigned long n) {
    unsigned long result=1;
    for (unsigned int i = 0; i < n; i++)
        result*=x;
    return result;
}

